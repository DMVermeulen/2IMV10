#include "hip/hip_runtime.h"
// compute.cu
#include <hip/hip_runtime.h>
#include<>
#include<math_functions.h>
#include<hip/hip_vector_types.h>
#include<vector_functions.h>
#include"hip/hip_vector_types.h"
#include<device_atomic_functions.h>
#include<hip/device_functions.h>
#include<sm_20_atomic_functions.h>
#include<sm_60_atomic_functions.h>

typedef unsigned int uint;

//Kernels 
//voxel count
__global__ void voxelCountKernel(
    float* oriTubesData,     //read from
    int* voxelCountData,     //write to
    int totalSize, int nVoxels_X, int nVoxels_Y, int nVoxels_Z, float3 aabbMin, float voxelUnitSize  //other parameters
) {
    int globalID = blockIdx.x * blockDim.x + threadIdx.x;

    if (globalID < totalSize) {
        float3 point = make_float3(oriTubesData[globalID * 3],
            oriTubesData[globalID * 3 + 1],
            oriTubesData[globalID * 3 + 2]);
        float3 deltaP = point - aabbMin;
        int level_X = min(nVoxels_X - 1, static_cast<int>(deltaP.x / voxelUnitSize));
        int level_Y = min(nVoxels_Y - 1, static_cast<int>(deltaP.y / voxelUnitSize));
        int level_Z = min(nVoxels_Z - 1, static_cast<int>(deltaP.z / voxelUnitSize));
        int index = nVoxels_X * nVoxels_Y * level_Z + nVoxels_X * level_Y + level_X;
        atomicAdd(&voxelCountData[index], 1);
    }
}

__global__ void densityEstimationKernal(
    int* voxelCountData,
    float* denseMapData,
    int totalSize, int nVoxels_X, int nVoxels_Y, int nVoxels_Z, int kernelR, float voxelUnitSize 
    ) {
    // Get global thread ID
    int globalID = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if global ID is within the valid range of the buffer
    if (globalID < totalSize) {
        float dense = 0;
        int Z = globalID / (nVoxels_X * nVoxels_Y);
        int mod = globalID % (nVoxels_X * nVoxels_Y);
        int X = mod % nVoxels_X;
        int Y = mod / nVoxels_X;
        for (int dx = -kernelR; dx < kernelR; dx++) {
            for (int dy = -kernelR; dy < kernelR; dy++) {
                for (int dz = -kernelR; dz < kernelR; dz++) {
                    int nx = X + dx;
                    nx = max(nx, 0);
                    nx = min(nx, nVoxels_X - 1);
                    int ny = Y + dy;
                    ny = max(ny, 0);
                    ny = min(ny, nVoxels_Y - 1);
                    int nz = Z + dz;
                    nz = max(nz, 0);
                    nz = min(nz, nVoxels_Z - 1);
                    float diff_x = voxelUnitSize * (nx - X);
                    float diff_y = voxelUnitSize * (ny - Y);
                    float diff_z = voxelUnitSize * (nz - Z);
                    float dot_product = diff_x * diff_x + diff_y * diff_y + diff_z * diff_z;
                    float PR2 = kernelR * voxelUnitSize * kernelR * voxelUnitSize;
                    if (dot_product > PR2)
                        continue;

                    int index = nVoxels_X * nVoxels_Y * nz + nVoxels_X * ny + nx;
                    index = min(index, totalSize);
                    int pointCnt = voxelCountData[index];

                    dense += pointCnt * (1 - dot_product / PR2);
                }
            }
        }

        denseMapData[globalID] = dense;
    }
}

__global__ void advectionKernel(
    float* oriTubesData, float* tempNormalsData, float* denseMapData,  //read from
    float* updatedTubesData,                       //write to
    int totalSize, int nVoxels_X, int nVoxels_Y, int nVoxels_Z, int kernelR, float voxelUnitSize, float3 aabbMin, int totalVoxels
 ) {
    int globalID = blockIdx.x * blockDim.x + threadIdx.x;

    if (globalID < totalSize) {
        float diff = 0;
        int R2 = kernelR * kernelR;
        int kernelWidth = 2 * kernelR + 1;
        float3 grad = make_float3(0);

        float3 point = make_float3(oriTubesData[globalID * 3],
            oriTubesData[globalID * 3 + 1],
            oriTubesData[globalID * 3 + 2]);
        float3 deltaP = point - aabbMin;
        int X = min(static_cast<int>(deltaP.x / voxelUnitSize), nVoxels_X - 1);
        int Y = min(static_cast<int>(deltaP.y / voxelUnitSize), nVoxels_Y - 1);
        int Z = min(static_cast<int>(deltaP.z / voxelUnitSize), nVoxels_Z - 1);
        int indexA = nVoxels_X * nVoxels_Y * Z + nVoxels_X * Y + X;

        for (int dx = -kernelR; dx < kernelR; dx++) {
            for (int dy = -kernelR; dy < kernelR; dy++) {
                for (int dz = -kernelR; dz < kernelR; dz++) {
                    int nx = X + dx;
                    nx = max(nx, 0);
                    nx = min(nx, nVoxels_X - 1);
                    int ny = Y + dy;
                    ny = max(ny, 0);
                    ny = min(ny, nVoxels_Y - 1);
                    int nz = Z + dz;
                    nz = max(nz, 0);
                    nz = min(nz, nVoxels_Z - 1);

                    float3 dir = make_float3(dx, dy, dz);
                    float diffPos = dot(dir, dir) / R2;
                    if (diffPos < 1e-5)
                        continue;
                    if (diffPos > 1)
                        continue;
                    int indexB = nVoxels_X * nVoxels_Y * nz + nVoxels_X * ny + nx;
                    indexB = min(indexB, totalVoxels);
                    float diffDense = denseMapData[indexB] - denseMapData[indexA];

                    if (diffDense > diff) {
                        grad = normalize(dir) * diffDense * exp(-diffPos);
                        diff = diffDense;
                    }
                }
            }
        }

        float3 delta;
        if (dot(grad, grad) < 1e-5)
            delta = make_float3(0);
        else
            delta = kernelR * voxelUnitSize * normalize(grad) / 1.5;

        //float3 normal = make_float3(tempNormalsData[globalID * 3],
        //    tempNormalsData[globalID * 3 + 1],
        //    tempNormalsData[globalID * 3 + 2]);
        //delta = dot(normal, delta) * normal;

        updatedTubesData[globalID * 3] = oriTubesData[globalID * 3] + delta.x;
        updatedTubesData[globalID * 3 + 1] = oriTubesData[globalID * 3 + 1] + delta.y;
        updatedTubesData[globalID * 3 + 2] = oriTubesData[globalID * 3 + 2] + delta.z;
    }
}

__global__ void relaxationKernel(
    float* tempTubesData, float* smoothedTubesData,
    float* relaxedTubesData,
    int totalSize, float relaxFactor
    ) {
    int globalID = blockIdx.x * blockDim.x + threadIdx.x;

    if (globalID < totalSize) {
        float3 newPoint = make_float3(smoothedTubesData[globalID * 3],
            smoothedTubesData[globalID * 3 + 1],
            smoothedTubesData[globalID * 3 + 2]);
        float3 oriPoint = make_float3(tempTubesData[globalID * 3],
            tempTubesData[globalID * 3 + 1],
            tempTubesData[globalID * 3 + 2]);

        float3 finalPoint = (1 - relaxFactor) * oriPoint + relaxFactor * newPoint;
        relaxedTubesData[globalID * 3] = finalPoint.x;
        relaxedTubesData[globalID * 3 + 1] = finalPoint.y;
        relaxedTubesData[globalID * 3 + 2] = finalPoint.z;
    }
}

//called by host 
extern "C" void cuda_voxelCount(float* oriTubesData,     //read from
    int* voxelCountData,    //write to
    int totalSize, int nVoxels_X, int nVoxels_Y, int nVoxels_Z, float3 aabbMin, float voxelUnitSize
) {
    int blockSize = 128; // Choose your desired block size
    int gridSize = (totalSize + blockSize - 1) / blockSize;
    voxelCountKernel << <gridSize, blockSize >> > (oriTubesData, voxelCountData, totalSize, nVoxels_X, nVoxels_Y, nVoxels_Z, aabbMin, voxelUnitSize);
}

extern "C" void cuda_densityEstimation(
    int* voxelCountData,      //read from
    float* denseMapData,      //write to
    int totalSize, int nVoxels_X, int nVoxels_Y, int nVoxels_Z, int kernelR, float voxelUnitSize
) {
    int blockSize = 128; // Choose your desired block size
    int gridSize = (totalSize + blockSize - 1) / blockSize;
    densityEstimationKernal << <gridSize, blockSize >> > (voxelCountData, denseMapData, totalSize, nVoxels_X, nVoxels_Y, nVoxels_Z, kernelR, voxelUnitSize);
}

extern "C" void cuda_advection(
    float* oriTubesData, float* tempNormalsData, float* denseMapData,  //read from
    float* updatedTubesData,                       //write to
    int totalSize, int nVoxels_X, int nVoxels_Y, int nVoxels_Z, int kernelR, float voxelUnitSize, float3 aabbMin, int totalVoxels
) {
    int blockSize = 128; // Choose your desired block size
    int gridSize = (totalSize + blockSize - 1) / blockSize;
    advectionKernel << <gridSize, blockSize >> > (oriTubesData, tempNormalsData, denseMapData, updatedTubesData, totalSize, nVoxels_X, nVoxels_Y, nVoxels_Z, kernelR, voxelUnitSize, aabbMin, totalVoxels);
}

extern "C" void cuda_relaxation(
    float* tempTubesData, float* smoothedTubesData,
    float* relaxedTubesData,
    int totalSize, float relaxFactor
) {
    int blockSize = 128; // Choose your desired block size
    int gridSize = (totalSize + blockSize - 1) / blockSize;
    relaxationKernel << <gridSize, blockSize >> > (tempTubesData, smoothedTubesData, relaxedTubesData, totalSize, relaxFactor);
}
